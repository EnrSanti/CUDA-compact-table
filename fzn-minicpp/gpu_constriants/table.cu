#include "hip/hip_runtime.h"
#include "gpu_constriants/table.cuh"
using namespace std;
using namespace Fca;
using namespace Gpu::Memory;

TableGPU::TableGPU(vector<var<int>::Ptr> & vars, vector<vector<int>> & tuples) : Table(vars,tuples){
    setPriority(CLOW);
    printf("%%%%%% TableGPU constructor\n");

    int noTuples=tuples.size();
    int noVars=vars.size();
    int currTableSize=(noTuples/32)+1;

    hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, 0);
    sm_count = device_prop.multiProcessorCount;
    int cores_per_SM = 128;
    printf("%%%%%% number of SMs: %d\n",sm_count);
    printf("%%%%%% warp size: %d\n",32);
    printf("%%%%%% cores per SM: %d\n",cores_per_SM);
    printf("%%%%%% support size: %d\n",_supportSize);
    
    
    // Memory allocation
    _currTable_dev = mallocDevice<unsigned int >(sizeof(unsigned int)*currTableSize); 
    _currTable_mask_dev = mallocDevice<unsigned int >(sizeof(unsigned int)*currTableSize); 
    _supports_dev = mallocDevice<unsigned int >(sizeof(unsigned int)*_supportSize*currTableSize);
    _supportSize_dev = mallocDevice<int>(sizeof(int));
    _variablesOffsets_dev = mallocDevice<int>(sizeof(int)*noVars);
    _supportOffsetJmp_dev = mallocDevice<int>(sizeof(int)*noVars);
    _currTable_size_dev=mallocDevice<int>(sizeof(int));
    _s_val_size_dev=mallocDevice<int>(sizeof(int));
    _s_val_dev=mallocDevice<int>(sizeof(int)*noVars);
    _vars_dev=mallocDevice<unsigned int>(sizeof(unsigned int)*((_supportSize/32)+1)); //matrix
    _currTable_reduction_dev=mallocDevice<unsigned int>(sizeof(unsigned int)*currTableSize*4); //matrix
    _output_dev=mallocDevice<int>(sizeof(int));
    printf("%%%%%% To store %d values i need %d words in my domains\n",_supportSize*currTableSize,((_supportSize/32)+1));
    
    //on host side we create simpler structures to then copy the data
    unsigned int *_currTable_host = mallocHost<unsigned int>(sizeof(unsigned int)*currTableSize); 
    unsigned int *_supports_host = mallocHost<unsigned int>(sizeof(unsigned int)*_supportSize*currTableSize);
    int *_vars_host=mallocHost<int>(sizeof(unsigned int)*((_supportSize/32)+1)); //matrix


    //get the vectors to arrays
    for(int i=0;i<_supportSize;i++){
        _supports_host[i*currTableSize]=_supports[i]._words.data()->value();
    }

    for(int i=0;i<((_supportSize/32)+1);i++){
        _vars_host[i]=0;
    }

    //can be done much better but for now it's ok
    for(int i=0;i<noVars;i++){
        vector<int> dom=_vars[i]->dumpDomainToVec();
        for(int j=0;j<dom.size();j++){
            //getting an unsigned int with the 32-dom[j]-_variablesOffsets[i] bit set
            unsigned int mask=1<<31-(dom[j]-_variablesOffsets[i]+_supportOffsetJmp[i]);
            int starting_word=(dom[j]-_variablesOffsets[i]+_supportOffsetJmp[i])/32;
            _vars_host[starting_word]=_vars_host[starting_word]|mask;
            //prinitng bits of _vars_host
        }
    }

    printf("%%%%%% %d\n",_vars_host[0]);
            
    //end of could be done better

    *_currTable_host=_currTable._words.data()->value();
    

    printf("%%%%%% check sizes 1: %lu %d \n",_currTable._words.size(),currTableSize);
    printf("%%%%%% check sizes 2: %lu %d \n",_supportSize*currTableSize,_supports.size()*_currTable._words.size());
    //printing currTable info
    printf("%%%%%% currTable: %u\n", _currTable._words.size());

    //Memory copy
    //hipMemcpyAsync(_supports_dev, _supports.data(), sizeof(SparseBitSet)*_supportSize, hipMemcpyHostToDevice);
    //(_currTable_dev, &_currTable, sizeof(SparseBitSet), hipMemcpyHostToDevice);
    hipMemcpyAsync(_supports_dev, _supports_host, sizeof(unsigned int)*_supportSize*currTableSize, hipMemcpyHostToDevice);
    hipMemcpyAsync(_currTable_dev, _currTable_host, sizeof(unsigned int)*currTableSize, hipMemcpyHostToDevice);
    hipMemcpyAsync(_supportSize_dev, &_supportSize, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpyAsync(_variablesOffsets_dev, _variablesOffsets.data(), sizeof(int)*noVars, hipMemcpyHostToDevice);
    hipMemcpyAsync(_supportOffsetJmp_dev, _supportOffsetJmp.data(), sizeof(int)*noVars, hipMemcpyHostToDevice);
    hipMemcpyAsync(_currTable_size_dev, &currTableSize, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpyAsync(_vars_dev, _vars_host, sizeof(unsigned int)*((_supportSize/32)+1), hipMemcpyHostToDevice);
    //printf("%%%%%% After mem cpy\n");
    //printGPUdata<<<1,1>>>(_supportSize_dev,_variablesOffsets_dev,_currTable_dev,_supports_dev,_supportOffsetJmp_dev,_currTable_size_dev);
    //hipDeviceSynchronize();
}
void TableGPU::post(){
    //printf("%%%%%% post GPU\n");
    for (auto const & v : _vars){
       v->propagateOnBoundChange(this);
    }
}
void TableGPU::propagate(){
    printf("%%%%%% propagate on GPU\n");
    enfoceGAC();
}

void TableGPU::enfoceGAC(){

    _s_val.clear();
    _s_sup.clear();
    
	for (int i = 0; i < _vars.size(); i++){
		//update s_val and the deltas
        //if(_vars[i]->changed()){
            _s_val.push_back(i);
        //}
        //update s_sup
        if(_vars[i]->size()>1){
            _s_sup.push_back(i);
        }
	}
    int size=_s_val.size();
    hipMemcpyAsync(_s_val_size_dev, &size, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpyAsync(_s_val_dev, _s_val.data(), sizeof(int)*size, hipMemcpyHostToDevice);

    printGPUdata<<<1,1>>>(_supportSize_dev,_variablesOffsets_dev,_currTable_dev,_supports_dev,_supportOffsetJmp_dev,_currTable_size_dev);
    hipDeviceSynchronize();
    
	updateTableGPU<<<4,32,130*sizeof(unsigned int)>>>(_supports_dev,_s_val_size_dev,_s_val_dev,_supportSize_dev,_variablesOffsets_dev,_supportOffsetJmp_dev,_currTable_dev,_currTable_size_dev,_vars_dev,_currTable_reduction_dev,_output_dev);
    hipDeviceSynchronize();
    int output;
    //retrieve the output from the device
    hipMemcpyAsync(&output, _output_dev, sizeof(int), hipMemcpyDeviceToHost);
    if(output==1){
        failNow();
        printf("%%%%%% fail now\n");
    }else{
        //we retrieve current table
    }
    
	//filterDomains();
}

// 1 th per support row
__global__ void updateTableGPU(unsigned int* _supports_dev,int * _s_val_size_dev, int *_s_val_dev, int *_supportSize_dev, int *_variablesOffsets_dev, int *_supportOffsetJmp_dev, unsigned int * _currTable_dev,int* _currTable_dev_size, unsigned int* _vars_dev, unsigned int* _currTable_reduction_dev, int* output){
  
    int thPos = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ unsigned int mask[]; //mask 
    int index=0;
    
    if(thPos>=*_supportSize_dev){
        return;
    }
    
    
    //clear mask TODO

    int varIndex=0;
    //every th checks which var it will be working on
    //TODO add check on the size of _s_val_dev oppure non mettere if(_vars[i]->changed()){ e invece che 3 metti _s_val_dev.size()
    for(int i=0; i<3; i++){
        if(thPos>=_supportOffsetJmp_dev[i]){
            varIndex++;
        }else{
            break;
        }
    }
    
    __syncthreads();

    index=_s_val_dev[varIndex]; //index tells me which var to access
    
    //check just _vars_dev[index]
    int index_x_a=thPos;
    //check if the bit at thPos of _vars_dev[index] is set to 1
    int word=thPos/32;
    int bit=thPos%32;
    unsigned int mask2=1<<31-bit;
    printf("%%%%%% i am thread %d and i am checking the bit %d of word %d of var %d, mask %u, the and value %d\n",thPos,bit,word,index,mask2, _vars_dev[word]&mask2);
    //RESET BASED UPDATE

    if ((_vars_dev[word]&mask2)!=0) {
        for(int i=0;i<*_currTable_dev_size;i++){
            atomicOr(&mask[i],_supports_dev[index_x_a*(*_currTable_dev_size)+i]);
            printf("%%%%%% accessing [%d] word of supports %d \n", index_x_a*(*_currTable_dev_size)+i, _supports_dev[index_x_a*(*_currTable_dev_size)+i]);
        }
    } 
    if(threadIdx.x==0){
        printf("%%%%%% th 0 of block %d\n",blockIdx.x);
        for(int i=0;i<*_currTable_dev_size;i++){
            _currTable_reduction_dev[blockIdx.x*(*_currTable_dev_size)+i]=mask[i];
        }
    }

    if(thPos==0){
        for(int i=0;i<*_currTable_dev_size;i++){
            for(int j=0; j<4; j++){
                mask[i]=mask[i] | _currTable_reduction_dev[j*(*_currTable_dev_size)+i];
            }
        }
        //_currTable.intersectWithMask();
        
        for(int i=0;i<*_currTable_dev_size;i++){
            _currTable_dev[i]=_currTable_dev[i] & mask[i];
            //delete printing 
            printf("\n%%%%%% ");
            int num=_currTable_dev[i];
            char str[32] = {'0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0'};
            for (int i = 31; i >= 0; i--) {
                str[i] = (num >> i) & 1; 
                printf("%d",str[i]);
            }
            printf("\n%%%%%% \n");
            //end delete
        }
        int empty=1;
        for(int i=0;i<*_currTable_dev_size;i++){
            if(_currTable_dev[i]!=0){
                *output=0;
                return;
            }
        }
        *output=1;
        printf("%%%%%% fail now GPU\n");
    }

}
//utilities
__global__ void printGPUdata(int *_supportSize_dev, int *_variablesOffsets_dev,unsigned int *_currTable_dev,unsigned int *_supports_dev,int * _supportOffsetJmp_dev, int* currTable_size_dev){
    printf("%%%%%% -------------------------- printGPUdata -------------------------- \n");
    printf("%%%%%% threadIdx.x: %d\n",threadIdx.x);
    printf("%%%%%% _supportSize_dev: %d\n",*_supportSize_dev);
    //printing the offsets
    printf("%%%%%% _variablesOffsets_dev: %d \n",_supportOffsetJmp_dev[0]);
    printf("%%%%%% _variablesOffsets_dev: %d \n",_supportOffsetJmp_dev[1]);
    printf("%%%%%% _variablesOffsets_dev: %d \n",_supportOffsetJmp_dev[2]);
    int k=0;
    int off=0;
    for(int i=0;i<*_supportSize_dev;i++){
        if(i==_supportOffsetJmp_dev[k]){
            printf("%%%%%% VAR %d\n",k);
            k++;
        }
        for(int j=0;j<*currTable_size_dev;j++){
            //we need to unwrap the bits
            printf("%%%%%% [%d] ", _variablesOffsets_dev[k]+i);
            printBitsGPU(_supports_dev[i*(*currTable_size_dev)+j]);
        }   
    }
    printf("%%%%%% currTable\n");
    for(int j=0;j<*currTable_size_dev;j++){
        printf("%%%%%% [%d] ", j);
        printBitsGPU(_currTable_dev[j]);
    }
 
}
void printBits(unsigned int num) {
    // Extracting each bit of the int and printing it
    //yes rather weird function, but since we need to print %%%%%
    char str[32] = {'0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0'};
    for (int i = 31; i >= 0; i--) {
        str[i] = (num >> i) & 1; 
        printf("%d",str[i]);
    }
    printf("\n%%%%%% \n");
}
__device__ void printBitsGPU(unsigned int num) {
    // Extracting each bit of the int and printing it
    //yes rather weird function, but since we need to print %%%%%
    char str[32] = {'0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0','0'};
    for (int i = 31; i >= 0; i--) {
        str[i] = (num >> i) & 1; 
        printf("%d",str[i]);
    }
    printf("\n%%%%%% \n");
}